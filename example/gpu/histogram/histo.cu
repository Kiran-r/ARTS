#include "hip/hip_runtime.h"
/******************************************************************************
** This material was prepared as an account of work sponsored by an agency   **
** of the United States Government.  Neither the United States Government    **
** nor the United States Department of Energy, nor Battelle, nor any of      **
** their employees, nor any jurisdiction or organization that has cooperated **
** in the development of these materials, makes any warranty, express or     **
** implied, or assumes any legal liability or responsibility for the accuracy,* 
** completeness, or usefulness or any information, apparatus, product,       **
** software, or process disclosed, or represents that its use would not      **
** infringe privately owned rights.                                          **
**                                                                           **
** Reference herein to any specific commercial product, process, or service  **
** by trade name, trademark, manufacturer, or otherwise does not necessarily **
** constitute or imply its endorsement, recommendation, or favoring by the   **
** United States Government or any agency thereof, or Battelle Memorial      **
** Institute. The views and opinions of authors expressed herein do not      **
** necessarily state or reflect those of the United States Government or     **
** any agency thereof.                                                       **
**                                                                           **
**                      PACIFIC NORTHWEST NATIONAL LABORATORY                **
**                                  operated by                              **
**                                    BATTELLE                               **
**                                     for the                               **
**                      UNITED STATES DEPARTMENT OF ENERGY                   **
**                         under Contract DE-AC05-76RL01830                  **
**                                                                           **
** Copyright 2019 Battelle Memorial Institute                                **
** Licensed under the Apache License, Version 2.0 (the "License");           **
** you may not use this file except in compliance with the License.          **
** You may obtain a copy of the License at                                   **
**                                                                           **
**    https://www.apache.org/licenses/LICENSE-2.0                            **
**                                                                           **
** Unless required by applicable law or agreed to in writing, software       **
** distributed under the License is distributed on an "AS IS" BASIS, WITHOUT **
** WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the  **
** License for the specific language governing permissions and limitations   **
******************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include "arts.h"
#include "artsGpuRuntime.h"
#include "hipblas.h"
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <assert.h>

#define ARRAYSIZE 65536
#define TILESIZE 8192
#define VERIFY 1
#define SMTILE 32 // Hardcoded for Volta
#define NUMBINS 10 // Make it a variable

uint64_t start = 0;

int inputArraySize;
int tileSize;
unsigned int numBlocks = 1;

artsGuid_t inputArrayGuid = NULL_GUID;
artsGuid_t histoGuid = NULL_GUID;
artsGuid_t doneGuid = NULL_GUID;
artsGuid_t finalSumGuid = NULL_GUID;

int * inputArray = NULL;
int * finalHistogram = NULL;

artsGuidRange * inputTileGuids = NULL;
artsGuidRange * partialHistoGuids = NULL;

__global__ void privateHistogram(uint32_t paramc, uint64_t * paramv, uint32_t depc, artsEdtDep_t depv[])
{
    const int numElements = (int) paramv[0];
    const int numBins = (int) paramv[1];
    int * tile = (int *) depv[0].ptr;
    int * localHisto = (int *) depv[1].ptr;

    // Compute histograms in every GPU
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x; //32
    int id = threadIdx.x;
    __shared__ unsigned int sbins[NUMBINS];

    // for (int i = id; i < num_bins; i += step)
    //     if (i < num_bins)
    //         sbins[i] = 0;
    // __syncthreads();
    step = blockDim.x * gridDim.x;// 0-8192 /32 => 0-31, 32-63... 
    for (int i = index; i < numElements; i += step)
        if (i < numElements)
            atomicAdd(&sbins[tile[i]], 1);
    __syncthreads();
    step = blockDim.x;
    for (int i = id; i < numBins; i += step)
    {
        if (i < numBins)
            atomicAdd(&localHisto[i], sbins[i]);
    }
}

__global__ void ReduceHistogram(uint32_t paramc, uint64_t * paramv, uint32_t depc, artsEdtDep_t depv[])
{
    // Reduce histograms from all GPUs into one.
    const int numBins = (int) paramv[0];
    const int numLocalHistograms = depc - 1;
    int * finalHisto = (int *) depv[0].ptr;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < numBins)
    {
        for (int i = 0; i < numLocalHistograms; i++)
        {
            int * localHisto = (int *) depv[1+i].ptr;
            finalHisto[index] += localHisto[index];
        }
    }
    assert(1);
}

void finishHistogram(uint32_t paramc, uint64_t * paramv, uint32_t depc, artsEdtDep_t depv[])
{
    uint64_t time = artsGetTimeStamp() - start;
    artsShutdown();
}

extern "C"
void initPerNode(unsigned int nodeId, int argc, char** argv)
{
    if (argc == 1)
    {
        inputArraySize = ARRAYSIZE;
        tileSize = TILESIZE;
    } else if (argc == 2)
    {
        inputArraySize = atoi(argv[1]);
        tileSize = TILESIZE;
    } else
    {
        inputArraySize = atoi(argv[1]);
        tileSize = atoi(argv[2]);
    }
    numBlocks = (inputArraySize+tileSize-1) / tileSize; // TODO: Fix if inputArraySize is < tileSize
    doneGuid = artsReserveGuidRoute(ARTS_EDT,            0);
    finalSumGuid = artsReserveGuidRoute(ARTS_GPU_EDT,    0);
    inputArrayGuid = artsReserveGuidRoute(ARTS_DB_READ,  0);
    histoGuid = artsReserveGuidRoute(ARTS_DB_WRITE,      0);

    inputTileGuids = artsNewGuidRangeNode(ARTS_DB_GPU_READ, numBlocks, 0);
    partialHistoGuids = artsNewGuidRangeNode(ARTS_DB_GPU_WRITE, numBlocks, 0);
    
    if (!nodeId)
    {
        inputArray = (int *) artsDbCreateWithGuid (inputArrayGuid, inputArraySize * sizeof(int));
        finalHistogram = (int *) artsDbCreateWithGuid (histoGuid, NUMBINS * sizeof(int));

        for (unsigned int elem = 0; elem < inputArraySize; elem++)
            inputArray[elem] = rand() % NUMBINS;
        
        for (unsigned int elem = 0; elem < NUMBINS; elem++)
            finalHistogram[elem] = 0;
        
        PRINTF("Starting...\n");
    }
}

extern "C"
void initPerWorker(unsigned int nodeId, unsigned int workerId, int argc, char** argv)
{
    unsigned int totalThreads = artsGetTotalNodes() * artsGetTotalWorkers();
    unsigned int globalThreadId = nodeId * artsGetTotalWorkers() + workerId;

    dim3 threads (SMTILE);
    dim3 grid((tileSize+SMTILE-1)/SMTILE);
    // dim3 grid(((tileSize*tileSize)+(SMTILE*SMTILE)-1)/(SMTILE*SMTILE), 1);

    if (!nodeId && !workerId)
    {
        for (unsigned int tile=0; tile<numBlocks; tile++)
        {
            artsGuid_t inputTileGuid = artsGetGuid(inputTileGuids, tile);
            artsGuid_t partialHistoGuid = artsGetGuid(partialHistoGuids, tile);
            int * inputTile = (int *) artsDbCreateWithGuid(inputTileGuid, sizeof(int) * tileSize);
            memcpy(inputTile, &inputArray[ tile * tileSize ], tileSize * sizeof(int));
            int * partialHisto = (int *) artsDbCreateWithGuid(partialHistoGuid, sizeof(int) *  NUMBINS);
            memset(partialHisto, 0, tileSize * sizeof(int));
        }
        uint64_t sumArgs[] = {tileSize};
        artsEdtCreateWithGuid (finishHistogram, doneGuid, 0, NULL, 1);
        artsEdtCreateGpuWithGuid (ReduceHistogram, finalSumGuid, 1, sumArgs, numBlocks+1, grid, threads, doneGuid, 0, histoGuid);
        artsSignalEdt(finalSumGuid, 0, histoGuid);
    }

    if (!workerId)
    {
        for (unsigned int tile=0; tile<numBlocks; tile++)
        {
            if (tile % artsGetTotalNodes() == nodeId)
            {
                artsGuid_t partialHistoGuid = artsGetGuid(partialHistoGuids, tile);
                uint64_t args[] = {numBlocks, tile};
                artsGuid_t privHistoGuid = artsEdtCreateGpu(privateHistogram, nodeId, 2, args, 2, grid, threads, finalSumGuid, 1+tile, partialHistoGuid);
                artsSignalEdt(privHistoGuid, 0, artsGetGuid(inputTileGuids, tile));
                artsSignalEdt(privHistoGuid, 1, partialHistoGuid);
            }
        }
    }
}

int main(int argc, char** argv)
{
    artsRT(argc, argv);
    return 0;
}